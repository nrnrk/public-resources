
#include <hip/hip_runtime.h>
extern "C" __global__ void count_wc(char *text, int *line_count, int *word_count, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;

    char c = text[idx];
    char prev = (idx == 0) ? ' ' : text[idx - 1];

    // Count lines
    if (c == '\n') {
        // Must be atomic because multiple threads can increment at the same time
        atomicAdd(line_count, 1);
    }

    // Count words (transition from ' ', '\n', or '\t' to character)
    if ((prev == ' ' || prev == '\n' || prev == '\t') && (c != ' ' && c != '\n' && c != '\t')) {
        atomicAdd(word_count, 1);
    }
}
